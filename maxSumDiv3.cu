#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <cassert>

// maxSumDivThreeCPU()为CPU实现版本，dp每次根据上一个位置的mod0、1、2的最大和更新当前位置mod0、1、2的最大子序列和。
// GPU版本实现思路：分为findMaxSumInSubArrays()和reduceModMaxsum()前后两部分：
// (为了方便实现，这里假设数组总长度能被(numBlocks*blockSize)整除)
// 1. findMaxSumInSubArrays()：首先把整个数组等分给每个block（假设1024个block），每个thread处理totalLength
// (numBlocks*blockSize)个data；每个thread的处理过程和CPU上的dp过程一致，for循环求得这部分数的mod0、1、2的最大子序列和；
// 于是通过findMaxSumInSubArrays()得到了numBlocks*blockSize组数的mod0、1、2的最大子序列和
// 2. reduceModMaxsum()：在每个block内采用reduce归并的方式更新这blockSize组数的mod0、1、2的最大子序列和
// 通过如下的方式对mod3=0，1，2的最大子序列和进行归并：
// 考虑两个数组a,b, 它们mod3=0，1，2的最大子序列和分别为(sumMod0_a,sumMod1_a,sumMod2_a),(sumMod0_b,sumMod1_b,sumMod2_b)
// 则a,b合并的数组c, mod3=0，1，2的最大子序列和为：(特别地，没有对应的mod和时记为负无穷)
// sumMod0_c = max(sumMod0_a+sumMod0_b, sumMod1_a+sumMod2_b, sumMod2_a+sumMod1_b)
// sumMod1_c = max(sumMod0_a+sumMod1_b, sumMod1_a+sumMod0_b, sumMod2_a+sumMod2_b)
// sumMod2_c = max(sumMod0_a+sumMod2_b, sumMod1_a+sumMod1_b, sumMod2_a+sumMod0_b)
// 在modMaxsum数组的第0/1/2个位置即为每个block对应数组mod 3 = 0/1/2的最大子序列和
// 最后在CPU上执行一次循环次数为numBlocks的最大子序列和合并，得到全局的最大子序列和

const long long NEGATIVE_INFINITY = -1e9;
__inline__ __device__ int deviceMax(int a, int b) {
    return a > b? a : b;
}


__global__ void findMaxSumInSubArrays(long long  *input, long long  *modMaxsum, int totalLength, int sub_array_length) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int g_tid = bid*blockDim.x+tid;
    int N = totalLength/gridDim.x; // N：data_preblock
    int n = N/blockDim.x; // n: data_prethread
    long long dp0 = 0, dp1 = 0, dp2 = 0;
    long long cur_dp0 = 0, cur_dp1 = 0, cur_dp2 = 0;

    // 每个thread处理n个data
    for (int i = 0; i < n; i++) {
        // 动态规划更新
        cur_dp0 = dp0;
        cur_dp1 = dp1;
        cur_dp2 = dp2;
        long long num = input[i + g_tid*n];       
        modMaxsum[g_tid*3] += num;
        // bug记录：数组动态分配内存函数提前退出，改为直接分配dp0,1,2
        {
           if((cur_dp0 + num) % 3 == 0){
               dp0 = max(dp0, cur_dp0 + num); 
           }
           else if((cur_dp0 + num) % 3 == 1){
               dp1= max(dp1, cur_dp0 + num); 
           }
           else{
               dp2= max(dp2, cur_dp0 + num); 
           }
        }
        {
           if((cur_dp1 + num) % 3 == 0){
               dp0 = max(dp0, cur_dp1 + num); 
           }
           else if((cur_dp1 + num) % 3 == 1){
               dp1= max(dp1, cur_dp1 + num); 
           }
           else{
               dp2= max(dp2, cur_dp1 + num); 
           }
        }
        {
           if((cur_dp2 + num) % 3 == 0){
               dp0 = max(dp0, cur_dp2 + num); 
           }
           else if((cur_dp2 + num) % 3 == 1){
               dp1= max(dp1, cur_dp2 + num); 
           }
           else{
               dp2= max(dp2, cur_dp2 + num); 
           }
        }
    }
    // modMaxsum记录每个thread统计的子数组的mod0、1、2的最大和
    modMaxsum[g_tid*3] = dp0;
    modMaxsum[g_tid*3+1] = dp1;
    modMaxsum[g_tid*3+2] = dp2;
}


__global__ void reduceModMaxsum(long long  *modMaxsum){
    __shared__ long long sdata[512*3]; //sdata长度为blockSize*3
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // 没有对应的mod和时记为负无穷
    sdata[tid*3] = modMaxsum[i*3]>0?modMaxsum[i*3]:NEGATIVE_INFINITY;
    sdata[tid*3+1] = modMaxsum[i*3+1]>0?modMaxsum[i*3+1]:NEGATIVE_INFINITY;
    sdata[tid*3+2] = modMaxsum[i*3+2]>0?modMaxsum[i*3+2]:NEGATIVE_INFINITY;
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            long long cur_modsum0 = sdata[tid*3],cur_modsum1 = sdata[tid*3+1],cur_modsum2 = sdata[tid*3+2];
            //printf("%d ", cur_modsum1);
            sdata[tid*3] = deviceMax(deviceMax(cur_modsum0+sdata[(tid+s)*3], cur_modsum1+sdata[(tid+s)*3+2]), cur_modsum2+sdata[(tid+s)*3+1]);
            sdata[tid*3+1] = deviceMax(deviceMax(cur_modsum0+sdata[(tid+s)*3+1], cur_modsum1+sdata[(tid+s)*3]), cur_modsum2+sdata[(tid+s)*3+2]);
            sdata[tid*3+2] = deviceMax(deviceMax(cur_modsum0+sdata[(tid+s)*3+2], cur_modsum2+sdata[(tid+s)*3]), cur_modsum1+sdata[(tid+s)*3+1]);
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        modMaxsum[blockIdx.x*3] = sdata[0];
        modMaxsum[blockIdx.x*3+1] = sdata[1];
        modMaxsum[blockIdx.x*3+2] = sdata[2];
    }        
}


// CPU实现版本
__host__ long long maxSumDivThreeCPU(long long *input, int length) {
    std::vector<long long > dp(3, 0);
    for (int i=0;i<length;i++) {
        long long num = input[i];
        std::vector<long long> current_dp = dp; 
        for (int sum : current_dp) {
            dp[(sum + num) % 3] = std::max(dp[(sum + num) % 3], sum + num);
        }
    }
    return dp[0]; 
}


int main() {
    int totalLength = 1024*1024*128;
    int numBlocks = 1024;
    int blockSize = 512;
    long long *h_input, *h_modMaxsum, *d_input, *d_modMaxsum;
    h_input = new long long[totalLength];
    h_modMaxsum = new long long[numBlocks*blockSize*3];

    // 随机初始化输入数据
    srand(3);
    for (int i = 0; i < totalLength; i++) {
        h_input[i] = static_cast<long long>(rand()%10+1);
    }

    hipMalloc((void**)&d_input, totalLength * sizeof(long long));
    hipMalloc((void**)&d_modMaxsum, numBlocks * blockSize * 3 * sizeof(long long));
    hipMemcpy(d_input, h_input, totalLength * sizeof(long long), hipMemcpyHostToDevice);
    assert(totalLength%(numBlocks*blockSize)==0);
    findMaxSumInSubArrays<<<numBlocks, blockSize>>>(d_input, d_modMaxsum, totalLength, totalLength/numBlocks/blockSize);
    // cudaMemcpy(h_modMaxsum, d_modMaxsum, numBlocks * blockSize * 3 * sizeof(int), cudaMemcpyDeviceToHost);
    // for(int i=0;i<10;i++)std::cout<< h_modMaxsum[i] << std::endl;

    // d_modMaxsum 存有numBlocks * blockSize 组{mod0maxsum,mod1maxsum,mod2maxsum}
    // 接下来采用reduce的方法两两合并modmaxsum
    reduceModMaxsum<<<numBlocks, blockSize>>>(d_modMaxsum);

    // 将每个block对应数组的最大和合并，得到全局的最大和
    // 将剩余的存有numBlocks组{mod0maxsum,mod1maxsum,mod2maxsum}循环求出最终的{mod0maxsum,mod1maxsum,mod2maxsum}
    hipMemcpy(h_modMaxsum, d_modMaxsum, numBlocks * blockSize * 3 * sizeof(long long), hipMemcpyDeviceToHost);
    std::vector<long long> modmaxsum = std::vector<long long>(3, 0);
    modmaxsum[0] = h_modMaxsum[0]>0?h_modMaxsum[0]:NEGATIVE_INFINITY;
    modmaxsum[1] = h_modMaxsum[1]>0?h_modMaxsum[1]:NEGATIVE_INFINITY;
    modmaxsum[2] = h_modMaxsum[2]>0?h_modMaxsum[2]:NEGATIVE_INFINITY;
    for(int i=1;i<numBlocks;i++){
        long long cur_modsum0 = modmaxsum[0],cur_modsum1 = modmaxsum[1],cur_modsum2 = modmaxsum[2];
        modmaxsum[0] = std::max(std::max(cur_modsum0+h_modMaxsum[i*3], cur_modsum1+h_modMaxsum[i*3+2]), cur_modsum2+h_modMaxsum[i*3+1]);
        modmaxsum[1] = std::max(std::max(cur_modsum0+h_modMaxsum[i*3+1], cur_modsum1+h_modMaxsum[i*3]), cur_modsum2+h_modMaxsum[i*3+2]);
        modmaxsum[2] = std::max(std::max(cur_modsum0+h_modMaxsum[i*3+2], cur_modsum2+h_modMaxsum[i*3]), cur_modsum1+h_modMaxsum[i*3+1]);
    }
    // for(int i=0;i<100;i++)std::cout<< h_modMaxsum[i] << std::endl;

    // 分别输出GPU和CPU实现的结果：
    std::cout << "GPU Result: " << modmaxsum[0]  << std::endl;
    std::cout << "CPU Result: " << maxSumDivThreeCPU(h_input,totalLength)  << std::endl;
    
    
    hipFree(d_input);
    hipFree(d_modMaxsum);
    delete[] h_input;
    delete[] h_modMaxsum;
    return 0;
}